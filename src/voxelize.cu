#include "hip/hip_runtime.h"
#include "voxelize.cuh"

// CUDA Global Memory variables
//__device__ size_t voxel_count = 0; // How many voxels did we count
//__device__ size_t triangles_seen_count = 0; // Sanity check

__constant__ uint32_t morton256_x[256];
__constant__ uint32_t morton256_y[256];
__constant__ uint32_t morton256_z[256];

// Encode morton code using LUT table
__device__ inline uint64_t mortonEncode_LUT(unsigned int x, unsigned int y, unsigned int z){
	uint64_t answer = 0;
	answer = morton256_z[(z >> 16) & 0xFF] |
		morton256_y[(y >> 16) & 0xFF] |
		morton256_x[(x >> 16) & 0xFF];
	answer = answer << 48 |
		morton256_z[(z >> 8) & 0xFF] |
		morton256_y[(y >> 8) & 0xFF] |
		morton256_x[(x >> 8) & 0xFF];
	answer = answer << 24 |
		morton256_z[(z)& 0xFF] |
		morton256_y[(y)& 0xFF] |
		morton256_x[(x)& 0xFF];
	return answer;
}

// Possible optimization: buffer bitsets (for now: Disabled because too much overhead)
//struct bufferedBitSetter{
//	unsigned int* voxel_table;
//	size_t current_int_location;
//	unsigned int current_mask;
//
//	__device__ __inline__ bufferedBitSetter(unsigned int* voxel_table, size_t index) :
//		voxel_table(voxel_table), current_mask(0) {
//		current_int_location = int(index / 32.0f);
//	}
//
//	__device__ __inline__ void setBit(size_t index){
//		size_t new_int_location = int(index / 32.0f);
//		if (current_int_location != new_int_location){
//			flush();
//			current_int_location = new_int_location;
//		}
//		unsigned int bit_pos = 31 - (unsigned int)(int(index) % 32);
//		current_mask = current_mask | (1 << bit_pos);
//	}
//
//	__device__ __inline__ void flush(){
//		if (current_mask != 0){
//			atomicOr(&(voxel_table[current_int_location]), current_mask);
//		}
//	}
//};

// Possible optimization: check bit before you set it - don't need to do atomic operation if it's already set to 1
// For now: overhead, so it seems
__device__ __inline__ bool checkBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	return ((voxel_table[int_location]) & (1 << bit_pos));
}
__device__ __inline__ void flipBit(unsigned int* voxel_table, size_t index) {
  size_t int_location = index / size_t(32);
  unsigned int bit_pos = size_t(31) - (index % size_t(32));
  unsigned int mask;
  mask = 1 << bit_pos;
  atomicXor(&(voxel_table[int_location]), mask);
}

// Set a bit in the giant voxel table. This involves doing an atomic operation on a 32-bit word in memory.
// Blocking other threads writing to it for a very short time
__device__ __inline__ void setBit(unsigned int* voxel_table, size_t index){
	size_t int_location = index / size_t(32);
	unsigned int bit_pos = size_t(31) - (index % size_t(32)); // we count bit positions RtL, but array indices LtR
	unsigned int mask = 1 << bit_pos;
	atomicOr(&(voxel_table[int_location]), mask);
}

__device__ __inline__ void unsetBit(unsigned int* voxel_table, size_t index) {

  size_t int_location = index / size_t(32);
  unsigned int bit_pos = size_t(31) - (index & size_t(32));
  unsigned int mask = 1 << bit_pos;
  atomicAnd(&(voxel_table[int_location]), ~mask);
}


// Main triangle voxelization method
__global__ void voxelize_triangle(voxinfo info, float* triangle_data, unsigned int* voxel_table, bool morton_order){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	// Common variables used in the voxelization process
	glm::vec3 delta_p(info.unit.x, info.unit.y, info.unit.z);
	glm::vec3 c(0.0f, 0.0f, 0.0f); // critical point
	glm::vec3 grid_max(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1); // grid max (grid runs from 0 to gridsize-1)

	while (thread_id < info.n_triangles){ // every thread works on specific triangles in its stride
		size_t t = thread_id * 9; // triangle contains 9 vertices

		// COMPUTE COMMON TRIANGLE PROPERTIES
		// Move vertices to origin using bbox
		glm::vec3 v0 = glm::vec3(triangle_data[t], triangle_data[t + 1], triangle_data[t + 2]) - info.bbox.min;
		glm::vec3 v1 = glm::vec3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - info.bbox.min; 
		glm::vec3 v2 = glm::vec3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - info.bbox.min;
		// Edge vectors
		glm::vec3 e0 = v1 - v0;
		glm::vec3 e1 = v2 - v1;
		glm::vec3 e2 = v0 - v2;
		// Normal vector pointing up from the triangle
		glm::vec3 n = glm::normalize(glm::cross(e0, e1));

		// COMPUTE TRIANGLE BBOX IN GRID
		// Triangle bounding box in world coordinates is min(v0,v1,v2) and max(v0,v1,v2)
		AABox<glm::vec3> t_bbox_world(glm::min(v0, glm::min(v1, v2)), glm::max(v0, glm::max(v1, v2)));
		// Triangle bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
		AABox<glm::ivec3> t_bbox_grid;
		t_bbox_grid.min = glm::clamp(t_bbox_world.min / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);
		t_bbox_grid.max = glm::clamp(t_bbox_world.max / info.unit, glm::vec3(0.0f, 0.0f, 0.0f), grid_max);

		// PREPARE PLANE TEST PROPERTIES
		if (n.x > 0.0f) { c.x = info.unit.x; }
		if (n.y > 0.0f) { c.y = info.unit.y; }
		if (n.z > 0.0f) { c.z = info.unit.z; }
		float d1 = glm::dot(n, (c - v0));
		float d2 = glm::dot(n, ((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		glm::vec2 n_xy_e0(-1.0f*e0.y, e0.x);
		glm::vec2 n_xy_e1(-1.0f*e1.y, e1.x);
		glm::vec2 n_xy_e2(-1.0f*e2.y, e2.x);
		if (n.z < 0.0f) {
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * glm::dot(n_xy_e0, glm::vec2(v0.x, v0.y))) + glm::max(0.0f, info.unit.x*n_xy_e0[0]) + glm::max(0.0f, info.unit.y*n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * glm::dot(n_xy_e1, glm::vec2(v1.x, v1.y))) + glm::max(0.0f, info.unit.x*n_xy_e1[0]) + glm::max(0.0f, info.unit.y*n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * glm::dot(n_xy_e2, glm::vec2(v2.x, v2.y))) + glm::max(0.0f, info.unit.x*n_xy_e2[0]) + glm::max(0.0f, info.unit.y*n_xy_e2[1]);
		// YZ plane
		glm::vec2 n_yz_e0(-1.0f*e0.z, e0.y);
		glm::vec2 n_yz_e1(-1.0f*e1.z, e1.y);
		glm::vec2 n_yz_e2(-1.0f*e2.z, e2.y);
		if (n.x < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * glm::dot(n_yz_e0, glm::vec2(v0.y, v0.z))) + glm::max(0.0f, info.unit.y*n_yz_e0[0]) + glm::max(0.0f, info.unit.z*n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * glm::dot(n_yz_e1, glm::vec2(v1.y, v1.z))) + glm::max(0.0f, info.unit.y*n_yz_e1[0]) + glm::max(0.0f, info.unit.z*n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * glm::dot(n_yz_e2, glm::vec2(v2.y, v2.z))) + glm::max(0.0f, info.unit.y*n_yz_e2[0]) + glm::max(0.0f, info.unit.z*n_yz_e2[1]);
		// ZX plane
		glm::vec2 n_zx_e0(-1.0f*e0.x, e0.z);
		glm::vec2 n_zx_e1(-1.0f*e1.x, e1.z);
		glm::vec2 n_zx_e2(-1.0f*e2.x, e2.z);
		if (n.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * glm::dot(n_zx_e0, glm::vec2(v0.z, v0.x))) + glm::max(0.0f, info.unit.x*n_zx_e0[0]) + glm::max(0.0f, info.unit.z*n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * glm::dot(n_zx_e1, glm::vec2(v1.z, v1.x))) + glm::max(0.0f, info.unit.x*n_zx_e1[0]) + glm::max(0.0f, info.unit.z*n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * glm::dot(n_zx_e2, glm::vec2(v2.z, v2.x))) + glm::max(0.0f, info.unit.x*n_zx_e2[0]) + glm::max(0.0f, info.unit.z*n_zx_e2[1]);

		// test possible grid boxes for overlap
		for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; z++){
			for (int y = t_bbox_grid.min.y; y <= t_bbox_grid.max.y; y++){
				for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; x++){
					// size_t location = x + (y*info.gridsize) + (z*info.gridsize*info.gridsize);
					// if (checkBit(voxel_table, location)){ continue; }

					// TRIANGLE PLANE THROUGH BOX TEST
					glm::vec3 p(x*info.unit.x, y*info.unit.y, z*info.unit.z);
					float nDOTp = glm::dot(n, p);
					if ((nDOTp + d1) * (nDOTp + d2) > 0.0f){ continue; }

					// PROJECTION TESTS
					// XY
					glm::vec2 p_xy(p.x, p.y);
					if ((glm::dot(n_xy_e0, p_xy) + d_xy_e0) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e1, p_xy) + d_xy_e1) < 0.0f){ continue; }
					if ((glm::dot(n_xy_e2, p_xy) + d_xy_e2) < 0.0f){ continue; }

					// YZ
					glm::vec2 p_yz(p.y, p.z);
					if ((glm::dot(n_yz_e0, p_yz) + d_yz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e1, p_yz) + d_yz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_yz_e2, p_yz) + d_yz_e2) < 0.0f){ continue; }

					// XZ	
					glm::vec2 p_zx(p.z, p.x);
					if ((glm::dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f){ continue; }

					//atomicAdd(&voxel_count, 1);
					if (morton_order){
						size_t location = mortonEncode_LUT(x, y, z);
						setBit(voxel_table, location);
					} else {
						size_t location = x + (y*info.gridsize.y) + (z*info.gridsize.y*info.gridsize.z);
						setBit(voxel_table, location);
					}
					continue;
				}
			}
		}
		// sanity check: atomically count triangles
		//atomicAdd(&triangles_seen_count, 1);
		thread_id += stride;
	}
}


__global__ void createPlane(voxinfo info, unsigned int* vtable) {
  int x = threadIdx.x;
  int z = blockIdx.x;

  size_t location = x + (10 * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
  setBit(vtable, location);
  location = x + (30 * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
  setBit(vtable, location);
}


__global__ void rasterization(voxinfo info, float* triangle_data, unsigned int* voxel_table, bool morton_order) {
  size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  size_t stride = blockDim.x * gridDim.x;

  glm::vec3 delta_p(info.unit.x, info.unit.y, info.unit.z);
  glm::vec3 grid_max(info.gridsize.x - 1, info.gridsize.y - 1, info.gridsize.z - 1);
  
  while (thread_id < info.n_triangles) {
    size_t t = thread_id * 9;

    glm::vec3 v0 = glm::vec3(triangle_data[t    ], triangle_data[t + 1], triangle_data[t + 2]) - info.bbox.min;
    glm::vec3 v1 = glm::vec3(triangle_data[t + 3], triangle_data[t + 4], triangle_data[t + 5]) - info.bbox.min;
    glm::vec3 v2 = glm::vec3(triangle_data[t + 6], triangle_data[t + 7], triangle_data[t + 8]) - info.bbox.min;

    glm::vec3 e0 = v1 - v0;
    glm::vec3 e1 = v2 - v1;
    glm::vec3 e2 = v0 - v2;

    glm::vec3 normal = glm::normalize(glm::cross(e0,e1));


    // COMPUTE TRIANGLE BOZ IN GRID
    AABox<glm::vec3> t_bbox_world(glm::min(v0, glm::min(v1, v2)), glm::max(v0, glm::max(v1, v2)));
    AABox<glm::ivec3> t_bbox_grid;
    t_bbox_grid.min = glm::clamp(t_bbox_world.min / info.unit, glm::vec3(0.0f), grid_max);
    t_bbox_grid.max = glm::clamp(t_bbox_world.max / info.unit, glm::vec3(0.0f), grid_max);

    // zx plane checks
		glm::vec2 n_zx_e0(-1.0f*e0.x, e0.z);
		glm::vec2 n_zx_e1(-1.0f*e1.x, e1.z);
		glm::vec2 n_zx_e2(-1.0f*e2.x, e2.z);
		if (normal.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * glm::dot(n_zx_e0, glm::vec2(v0.z, v0.x)));
		float d_xz_e1 = (-1.0f * glm::dot(n_zx_e1, glm::vec2(v1.z, v1.x)));
		float d_xz_e2 = (-1.0f * glm::dot(n_zx_e2, glm::vec2(v2.z, v2.x)));

    for (int z = t_bbox_grid.min.z; z <= t_bbox_grid.max.z; ++z) {
      for (int x = t_bbox_grid.min.x; x <= t_bbox_grid.max.x; ++x) {
        for (int y = t_bbox_grid.min.y; y <= grid_max.y; ++y) {
          glm::vec3 voxel_centre(x * info.unit.x + info.unit.x / 2.0f,
                                 y * info.unit.y + info.unit.y / 2.0f,
                                 z * info.unit.z + info.unit.z / 2.0f);
          
					// XZ	
					glm::vec2 p_zx(voxel_centre.z, voxel_centre.x);
					if ((glm::dot(n_zx_e0, p_zx) + d_xz_e0) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e1, p_zx) + d_xz_e1) < 0.0f){ continue; }
					if ((glm::dot(n_zx_e2, p_zx) + d_xz_e2) < 0.0f){ continue; }

          size_t location; 
          if (morton_order) {
            location = mortonEncode_LUT(x, y, z);
          } else {
            location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
          }

          if (checkVoxel(x, y - 1, z, info.gridsize.x, voxel_table) == char(1)) 
            continue;

          setBit(voxel_table, location);
          break;
        }
      }
    }

  thread_id += stride;
  }
}

//__global__ void fill_pass(voxinfo info, float* triangle_data, unsigned int* vtable, bool morton_order) {
//  int x = threadIdx.x;
//  int z = blockIdx.x;
//
//  int ymax = info.gridsize.y - 1;
//
//  bool flip = false;
//  for (int y = 1; y < ymax; ++y) {
//    char value = checkVoxel(x, y, z, info.gridsize.x, vtable);
//     flip = true;
//
//    
//    size_t location;
//    if (morton_order) {
//      location = mortonEncode_LUT(x, y, z);
//    } else {
//      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
//    }
//
//    size_t int_location = location / size_t(32);
//    unsigned int bit_pos = size_t(31) - (location % size_t(32));
//
//    unsigned int mask = 0;
//    if (prev_value == char(1)) {
//      mask = 1 << bit_pos;
//    } 
//    atomicXor(&(vtable[int_location]), mask);
//  }
//}


__global__ void carveY(unsigned int* vtable_obj, unsigned int* vtable_carved, 
                       const voxinfo& info, bool morton_order) {
  int x = threadIdx.x;
  int z = blockIdx.x;

  size_t ymax = info.gridsize.y - 1;

  for (size_t y = 0; y < ymax; ++y) {
    size_t location;
    if (morton_order) {
      location = mortonEncode_LUT(x, y, z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) { 
      break;
    }

    unsetBit(vtable_carved, location);
  }

  for (size_t y = ymax; y >= 0; --y) {
    size_t location;
    if (morton_order) {
      location = mortonEncode_LUT(x, y, z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) {
      break;
    }


    unsetBit(vtable_carved, location);
  }
}


__global__ void carveX(unsigned int* vtable_obj, unsigned int* vtable_carved, 
                      const voxinfo& info, bool morton_order) {
  int y = threadIdx.x;
  int z = blockIdx.x;

  size_t xmax = info.gridsize.x - 1;
  
  for (size_t x = 0; x <= xmax; ++x) {
    size_t location;

    if (morton_order) {
      location = mortonEncode_LUT(x, y, z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) {
      break;
    }


    unsetBit(vtable_carved, location);
  }

  for (size_t x = xmax; x >= 0; --x) {

    size_t location;

    if (morton_order) {
      location = mortonEncode_LUT(x, y ,z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) {
      break;
    }

    unsetBit(vtable_carved, location);
  }
}


__global__ void carveZ(unsigned int* vtable_obj, unsigned int* vtable_carved, 
                       const voxinfo& info, bool morton_order) {
  int x = threadIdx.x;
  int y = blockIdx.x;

  size_t zmax = info.gridsize.z - 1;

  for(size_t z = 0; z <= zmax; ++z) {
    size_t location;

    if(morton_order) {
      location = mortonEncode_LUT(x, y, z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) {
      break;
    }


    unsetBit(vtable_carved, location);
  }

  for (size_t z = zmax; z >= 0; --z) {

    size_t location;

    if(morton_order) {
      location = mortonEncode_LUT(x, y, z);
    } else {
      location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
    }

    if (checkBit(vtable_obj, location)) {
      break;
    }

    unsetBit(vtable_carved, location);
  }
}

__global__ void mergeVtables(unsigned int* v1, unsigned int* v2, const voxinfo& info, bool morton_order) {
  int x = threadIdx.x;
  printf("%d\n", x);

  size_t zmax = info.gridsize.z - 1;
  size_t ymax = info.gridsize.y - 1;

  for (size_t y = 0; y <= ymax; ++y) {

    for (size_t z = 0; z <= zmax; ++z) {

      size_t location;
      if(morton_order) {
        location = mortonEncode_LUT(x, y, z);
      } else {
        location = x + (y * info.gridsize.y) + (z * info.gridsize.y * info.gridsize.z);
      }

      printf("working");
      if (!checkBit(v2, location)) {
        continue;
      }

      setBit(v1, location);
    }
  }
}

void solid_voxelize(const voxinfo& v, std::vector<float*> triangle_data, unsigned int* vtable, 
    bool morton_code) {
  float elapsed_time;


  if (morton_code) {
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_x), host_morton256_x, 256 * sizeof(uint32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_y), host_morton256_y, 256 * sizeof(uint32_t)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_z), host_morton256_z, 256 * sizeof(uint32_t)));
  }

  int blockSize;
  int minGridSize;
  int gridSize;

  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle, 0, 0);
  gridSize = (v.n_triangles + blockSize - 1) / blockSize;
  std::cout << "CUDA GRIDSIZE: " << gridSize << " , " << "BLOCKSIZE: " << blockSize << std::endl;

  size_t vtable_size = ((size_t) v.gridsize.x * v.gridsize.y * v.gridsize.z) / 8.0f;

  unsigned int* vtable_carved;
  checkCudaErrors(hipMallocManaged((void**)&vtable_carved, vtable_size));
  unsigned int* vtable_obj;
  checkCudaErrors(hipMallocManaged((void**)&vtable_obj, vtable_size));

  int count = 1;
  for (float* gpu_triangles: triangle_data) {
    std::cout << " - voxelizing obj #  " << count << std::endl;
//    checkCudaErrors(hipMemset((void**)vtable_obj, -1, vtable_size));
//    checkCudaErrors(hipMemset((void**)vtable_obj, -1, vtable_size));

    voxelize_triangle<<<gridSize, blockSize>>>(v, gpu_triangles, vtable_obj, morton_code);
    //carveX<<<v.gridsize.y, v.gridsize.z>>>(vtable_obj, vtable_carved, v, morton_code);
    //carveY<<<v.gridsize.x, v.gridsize.z>>>(vtable_obj, vtable_carved, v, morton_code);
    //carveZ<<<v.gridsize.x, v.gridsize.y>>>(vtable_obj, vtable_carved, v, morton_code);
    //mergeVtables<<<v.gridsize.x, v.gridsize.y>>>(vtable_obj, vtable_carved, v, morton_code);
    mergeVtables<<<1, 256>>>(vtable, vtable_obj, v, morton_code);
    hipDeviceSynchronize();
    ++count;
  }
  // checkCudaErrors(hipMemcpy(vtable, vtable_obj, vtable_size, hipMemcpyDeviceToDevice));
 

  //vtable = vtable_obj;
}



void voxelize(const voxinfo& v, float* triangle_data, unsigned int* vtable, 
    bool useThrustPath, bool morton_code, bool solid) {
	float   elapsedTime;

	// These are only used when we're not using UNIFIED memory
	unsigned int* dev_vtable; // DEVICE pointer to voxel_data
	size_t vtable_size; // vtable size
	
	// Create timers, set start time
	hipEvent_t start_vox, stop_vox;
	checkCudaErrors(hipEventCreate(&start_vox));
	checkCudaErrors(hipEventCreate(&stop_vox));

	// Copy morton LUT if we're encoding to morton
	if (morton_code){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_x), host_morton256_x, 256 * sizeof(uint32_t)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_y), host_morton256_y, 256 * sizeof(uint32_t)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(morton256_z), host_morton256_z, 256 * sizeof(uint32_t)));
	}

	// Estimate best block and grid size using CUDA Occupancy Calculator
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the  maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, voxelize_triangle, 0, 0);
	// Round up according to array size 
	gridSize = (v.n_triangles + blockSize - 1) / blockSize;

	if (useThrustPath) { // We're not using UNIFIED memory
		vtable_size = ((size_t)v.gridsize.x * v.gridsize.y * v.gridsize.z) / (size_t) 8.0;
		fprintf(stdout, "[Voxel Grid] Allocating %llu kB of DEVICE memory\n", size_t(vtable_size / 1024.0f));
		checkCudaErrors(hipMalloc(&dev_vtable, vtable_size));
		checkCudaErrors(hipMemset(dev_vtable, 0, vtable_size));
		// Start voxelization
		checkCudaErrors(hipEventRecord(start_vox, 0));
		voxelize_triangle << <gridSize, blockSize >> > (v, triangle_data, dev_vtable, morton_code);
	}
	else { // UNIFIED MEMORY 
		checkCudaErrors(hipEventRecord(start_vox, 0));
		voxelize_triangle << <gridSize, blockSize >> > (v, triangle_data, vtable, morton_code);
	}

	hipDeviceSynchronize();
	checkCudaErrors(hipEventRecord(stop_vox, 0));
	checkCudaErrors(hipEventSynchronize(stop_vox));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
	printf("[Voxelization] GPU time:  %3.1f ms\n", elapsedTime);

	// If we're not using UNIFIED memory, copy the voxel table back and free all
	if (useThrustPath){
		fprintf(stdout, "[Voxel Grid] Copying %llu kB to page-locked HOST memory\n", size_t(vtable_size / 1024.0f));
		checkCudaErrors(hipMemcpy((void*)vtable, dev_vtable, vtable_size, hipMemcpyDefault));
		fprintf(stdout, "[Voxel Grid] Freeing %llu kB of DEVICE memory\n", size_t(vtable_size / 1024.0f));
		checkCudaErrors(hipFree(dev_vtable));
	}

	// SANITY CHECKS
	//size_t t_seen, v_count;
	//HANDLE_CUDA_ERROR(hipMemcpyFromSymbol((void*)&(t_seen),triangles_seen_count, sizeof(t_seen), 0, hipMemcpyDeviceToHost));
	//HANDLE_CUDA_ERROR(hipMemcpyFromSymbol((void*)&(v_count), voxel_count, sizeof(v_count), 0, hipMemcpyDeviceToHost));
	//printf("We've seen %llu triangles on the GPU \n", t_seen);
	//printf("We've found %llu voxels on the GPU \n", v_count);

	// Destroy timers
	checkCudaErrors(hipEventDestroy(start_vox));
	checkCudaErrors(hipEventDestroy(stop_vox));
}
